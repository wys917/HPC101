#include "hip/hip_runtime.h"

#include "conv.cuh"

#define a(n, h, w, c) a[(n) * H * W * C + (h) * W * C + (w) * C + (c)]
#define w(k, r, s, c) w[(k) * R * S * C + (r) * S * C + (s) * C + (c)]
#define b(n, h, w, k) b[(n) * H * W * K + (h) * W * K + (w) * K + (k)]

static constexpr int BLOCK = 16;
static constexpr int BLOCK_H = BLOCK;
static constexpr int BLOCK_W = BLOCK;

// =======================================================
// --- INT8 版本 ---
// =======================================================
template <>
__global__ void conv2d_cuda_kernel<int8_t, int>(const int8_t *__restrict__ a,
                                                const int8_t *__restrict__ w,
                                                int8_t *__restrict__ b) {
    extern __shared__ int8_t s_mem_int8[];                 
    const int INPUT_TILE_SIZE_IN_ELEMENTS = (BLOCK_H + R - 1) * (BLOCK_W + S - 1);
    int8_t* s_a = s_mem_int8;                          
    int8_t* s_w = s_mem_int8 + INPUT_TILE_SIZE_IN_ELEMENTS; 

    // === 1. 坐标与ID计算 ===
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * BLOCK_W + tx;
    const int tile_h_start = blockIdx.y * BLOCK_H;
    const int tile_w_start = blockIdx.x * BLOCK_W;
    const int output_h = tile_h_start + ty;
    const int output_w = tile_w_start + tx;
    const int NUM_THREADS = BLOCK_H * BLOCK_W;

    // === 2. 主计算循环 ===
    for (int n = 0; n < N; ++n) {
        for (int k = 0; k < K; ++k) {
            int accumulator = 0; 

            for (int c = 0; c < C; ++c) {
                
                const int INPUT_TILE_H = BLOCK_H + R - 1;
                const int INPUT_TILE_W = BLOCK_W + S - 1;
                for (int i = tid; i < INPUT_TILE_SIZE_IN_ELEMENTS; i += NUM_THREADS) {
                    const int smem_h = i / INPUT_TILE_W;
                    const int smem_w = i % INPUT_TILE_W;
                    const int g_ih = tile_h_start + smem_h - R / 2;
                    const int g_iw = tile_w_start + smem_w - S / 2;
                    if (g_ih >= 0 && g_ih < H && g_iw >= 0 && g_iw < W) {
                        s_a[i] = a(n, g_ih, g_iw, c);
                    } else {
                        s_a[i] = static_cast<int8_t>(0);
                    }
                }
                const int WEIGHT_SIZE = R * S;
                for (int i = tid; i < WEIGHT_SIZE; i += NUM_THREADS) {
                    const int r = i / S;        
                    const int s = i % S;     
                    s_w[i] = w(k, r, s, c);
                }

                
                __syncthreads();

           
                if (output_h < H && output_w < W) {
                    for (int r = 0; r < R; ++r) {
                        for (int s = 0; s < S; ++s) {
                            int8_t s_val = s_a[(ty + r) * INPUT_TILE_W + (tx + s)];
                            int8_t w_val = s_w[r * S + s];
                            accumulator += static_cast<int>(s_val) * static_cast<int>(w_val);
                        }
                    }
                }

               
                __syncthreads();

            } 

           
            if (output_h < H && output_w < W) {
                b(n, output_h, output_w, k) = static_cast<int8_t>(accumulator);
            }
        } 
    } 
}

// =======================================================
// --- HALF 版本 ---
// =======================================================
template <>
__global__ void conv2d_cuda_kernel<half_t, float>(const half_t *__restrict__ a,
                                                  const half_t *__restrict__ w,
                                                  half_t *__restrict__ b) {
    
    extern __shared__ half_t s_mem_half[];                
    const int INPUT_TILE_SIZE_IN_ELEMENTS = (BLOCK_H + R - 1) * (BLOCK_W + S - 1);
    half_t* s_a = s_mem_half;                               
    half_t* s_w = s_mem_half + INPUT_TILE_SIZE_IN_ELEMENTS; 

    // === 1. 坐标与ID计算 ===
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * BLOCK_W + tx;
    const int tile_h_start = blockIdx.y * BLOCK_H;
    const int tile_w_start = blockIdx.x * BLOCK_W;
    const int output_h = tile_h_start + ty;
    const int output_w = tile_w_start + tx;
    const int NUM_THREADS = BLOCK_H * BLOCK_W;

    // === 2. 主计算循环 ===
    for (int n = 0; n < N; ++n) {
        for (int k = 0; k < K; ++k) {
            float accumulator = 0.0f; 

            for (int c = 0; c < C; ++c) {
      
                const int INPUT_TILE_H = BLOCK_H + R - 1;
                const int INPUT_TILE_W = BLOCK_W + S - 1;
                for (int i = tid; i < INPUT_TILE_SIZE_IN_ELEMENTS; i += NUM_THREADS) {
                    const int smem_h = i / INPUT_TILE_W;
                    const int smem_w = i % INPUT_TILE_W;
                    const int g_ih = tile_h_start + smem_h - R / 2;
                    const int g_iw = tile_w_start + smem_w - S / 2;
                    if (g_ih >= 0 && g_ih < H && g_iw >= 0 && g_iw < W) {
                        s_a[i] = a(n, g_ih, g_iw, c);
                    } else {
                        s_a[i] = static_cast<half_t>(0.0f);
                    }
                }
                const int WEIGHT_SIZE = R * S;
                for (int i = tid; i < WEIGHT_SIZE; i += NUM_THREADS) {
                    const int r = i / S;      
                    const int s = i % S;       
                    s_w[i] = w(k, r, s, c);
                }
             
                __syncthreads();

                if (output_h < H && output_w < W) {
                    for (int r = 0; r < R; ++r) {
                        for (int s = 0; s < S; ++s) {
                            half_t s_val = s_a[(ty + r) * INPUT_TILE_W + (tx + s)];
                            half_t w_val = s_w[r * S + s];
                            
                            accumulator += static_cast<float>(s_val) * static_cast<float>(w_val);
                        }
                    }
                }
                
             
                __syncthreads();

            } 

          
            if (output_h < H && output_w < W) {
                b(n, output_h, output_w, k) = static_cast<half_t>(accumulator);
            }
        } 
    } 
}

// =======================================================
// --- 配置函数  ---
// =======================================================
template <>
KernelConfig get_kernel_config<int8_t>() {
    KernelConfig config;
    config.grid = dim3((W + BLOCK_W - 1) / BLOCK_W, (H + BLOCK_H - 1) / BLOCK_H);
    config.block = dim3(BLOCK_W, BLOCK_H, 1);
    const int input_tile_size = (BLOCK_H + R - 1) * (BLOCK_W + S - 1) * sizeof(int8_t);
    const int weight_tile_size = R * S * sizeof(int8_t);
    config.shared_memory_size = input_tile_size + weight_tile_size;
    return config;
}

template <>
KernelConfig get_kernel_config<half_t>() {
    KernelConfig config;
    config.grid = dim3((W + BLOCK_W - 1) / BLOCK_W, (H + BLOCK_H - 1) / BLOCK_H);
    config.block = dim3(BLOCK_W, BLOCK_H, 1);
    const int input_tile_size = (BLOCK_H + R - 1) * (BLOCK_W + S - 1) * sizeof(half_t);
    const int weight_tile_size = R * S * sizeof(half_t);
    config.shared_memory_size = input_tile_size + weight_tile_size;
    return config;
}